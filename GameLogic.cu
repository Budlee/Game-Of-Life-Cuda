#include "hip/hip_runtime.h"
#include "GameLogic.h"


void GameLogic(int64_t xIn, int64_t yIn)
{
	processorType = CPU_PROCESSOR;
    x = xIn;
    y = yIn;
    cellSwitch = 0;
    int64_t totalBlockCount = x*y;
    cells = (uint8_t*)malloc(sizeof (uint8_t)*(totalBlockCount));
    outputCell = (uint8_t*)malloc(sizeof (uint8_t)*(totalBlockCount));
    srand(time(NULL));
    while(totalBlockCount-- != 0)
    {
        if(rand()%2)
        {
            cells[totalBlockCount] = 1;
        }
        else
        {
        	cells[totalBlockCount] = 0;
        }
        outputCell[totalBlockCount]= 0;
    }
    cudaSetup();
}

void desructor_GameLogic()
{
	printf("\n\nEXIT\n\n");
	hipFree(d_data_in);
	hipFree(d_data_in_x);
	hipFree(d_data_in_y);
	hipFree(d_data_out);
    free(cells);
    free(outputCell);
}

uint8_t* getGameOfLifeState()
{
    return cells;
}

void processWithCPU(){processorType = CPU_PROCESSOR;}
void processWithGPUBasic(){processorType = GPU_BASIC_PROCESSOR;}
void processWithGPUOpt(){processorType = GPU_OPTIMIZED_PROCESSOR;}

void step()
{
	if(processorType == CPU_PROCESSOR)
	{
		cpuImplementation();
	}
	else
	{
		GPUImplementation();
	}
}

void cpuImplementation()
{
    uint64_t yId =0, xId = 0;
    while(yId < y)
    {
        xId = 0;
        while(xId < x)
        {
        	outputCell[xId+(yId*y)] = 0;
            uint8_t localCellCount =  surrondingCellCount(cells, xId, yId, x, y);
            switch(localCellCount)
            {
                case 2:
                    if(cells[xId+(yId*y)])
                    {
                    	outputCell[xId+(yId*y)] = 1;
                    }
                    break;
                case 3:
                	outputCell[xId+(yId*y)] = 1;
                    break;
            }
            ++xId;
        }
        ++yId;
    }
    uint8_t * switcher = cells;
    cells = outputCell;
    outputCell = switcher;
}

__host__ __device__ uint8_t surrondingCellCount(uint8_t *cellsLocal, int64_t xCell, int64_t yCell, int64_t x, int64_t y)
{
    //Parse left -> right, top -> bottom
    uint8_t cellCount = 0;
    xCell= xCellMinus(1,xCell, x);
    yCell = yCellMinus(1,yCell, y);
    uint8_t searchIndex = 0;
    while(searchIndex < 9)
    {
        if(searchIndex == 4)
        {
        	xCell = xCellPlus(1,xCell, x);
            ++searchIndex;
            continue;
        }
        if(searchIndex == 3 || searchIndex == 6)
        {
        	xCell =xCellMinus(3,xCell, x);
        	yCell = yCellPlus(1,yCell, y);
        }
        //find cell
        int64_t cell = xCell + (y * yCell);
        if(cellsLocal[cell])
        {
            cellCount++;
        }

        xCell = xCellPlus(1,xCell, x);
        ++searchIndex;
    }

    return cellCount;
}

__device__ uint8_t surrondingCellCountOptimized(uint8_t *cellsLocal, int64_t xCell, int64_t x, int64_t y)
{
	//Parse left -> right, top -> bottom
	    uint8_t cellCount = 0;
	    int64_t yCell = 1;
	    xCell= xCellMinus(1,xCell, x);
	    yCell = yCellMinus(1,yCell, y);
	    uint8_t searchIndex = 0;
	    while(searchIndex < 9)
	    {
	        if(searchIndex == 4)
	        {
	        	xCell = xCellPlus(1,xCell, x);
	            ++searchIndex;
	            continue;
	        }
	        if(searchIndex == 3 || searchIndex == 6)
	        {
	        	xCell =xCellMinus(3,xCell, x);
	        	yCell = yCellPlus(1,yCell, y);
	        }
	        //find cell
	        int64_t cell = xCell + (y * yCell);
	        if(cellsLocal[cell])
	        {
	            cellCount++;
	        }
	        if(cellCount > 3)
	        {
	        	return cellCount;
	        }

	        xCell = xCellPlus(1,xCell, x);
	        ++searchIndex;
	    }

	    return cellCount;

}

__host__ __device__ int64_t xCellPlus(uint8_t add, int64_t value, int64_t x)
{
    if(add == 0 && add < x)
    {
        return -1;
    }
    if(value+add >= x)
    {
        value=(value+add) - x ;
    }
    else
    {
        value+=add;
    }
    return value;
}

__host__ __device__ int64_t xCellMinus(uint8_t minus, int64_t value, int64_t x)
{
    if(minus == 0 && minus < x)
    {
        return -1;
    }
    if(value-minus < 0)
    {
        value = x + (value-minus);
    }
    else
    {
        value -= minus;
    }
    return value;
}

__host__ __device__ int64_t yCellPlus(uint8_t add, int64_t value, int64_t y)
{
    if(add == 0 && add < y)
    {
        return -1;
    }
    if(value+add >= y)
    {
        value=(value+add) - y;
    }
    else
    {
        value+=add;
    }
    return value;
}

__host__ __device__ int64_t yCellMinus(uint8_t minus, int64_t value, int64_t y)
{
    if(minus == 0 && minus < y)
    {
        return -1;
    }
    if(value-minus < 0)
    {
        value = y +(value-minus);
    }
    else
    {
        value -= minus;
    }
    return value;
}

void cudaSetup()
{
    printf("Get GPU properties\n");
    int64_t totalBlockCount = x*y;
	blockSize = 1024;
	gridSize = ((totalBlockCount + (blockSize-1))/blockSize);
	optBlockSize = x;
	optGridSize =(((x*y) + (optBlockSize-1))/optBlockSize);
	if((x >= 1024) || (y >= 1024))
	{
		printf("\n\nUnable to use optimized GPU version with x & y larger than 1023\n\n");
	}

    int devID;
    hipDeviceProp_t props;
    hipGetDevice(&devID);
    hipGetDeviceProperties(&props, devID);
    printf("Allocate memory on GPU\n");
    hipMalloc((void**) &d_data_in, (sizeof(uint8_t) * totalBlockCount));
    hipMalloc((void**) &d_data_in_x, (sizeof(int64_t)));
    hipMalloc((void**) &d_data_in_y, (sizeof(int64_t)));
    hipMalloc((void**) &d_data_out, (sizeof(uint8_t) * totalBlockCount));

}

void GPUImplementation()
{
	//Copy data to GPU
	int64_t totalBlockCount = x*y;
    hipMemcpy(d_data_in, cells, (sizeof(uint8_t) * totalBlockCount), hipMemcpyHostToDevice);
    hipMemcpy(d_data_in_x, &x, sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_data_in_y, &y, sizeof(int64_t), hipMemcpyHostToDevice);
    //Executing GPU calculation
    if(processorType == GPU_OPTIMIZED_PROCESSOR && (x < 1024) && (y < 1024))
    {
		gpuGameOfLifeOptimized<<<optGridSize, optBlockSize, sizeof(uint8_t)*(x * 3)>>>(d_data_in, d_data_in_x, d_data_in_y, d_data_out);
    }
    else
    {
    	gpuGameOfLifeNaive<<<gridSize, blockSize>>>(d_data_in, d_data_in_x, d_data_in_y, d_data_out);
    }
    hipDeviceSynchronize();
    //Complete GPU calculation
    //Reading GPU data out
    hipMemcpy(outputCell, d_data_out, (sizeof(uint8_t) * totalBlockCount), hipMemcpyDeviceToHost);
    uint8_t * switcher = cells;
	cells = outputCell;
	outputCell = switcher;
}
/**
 * This optimization does not work... Also this has to be for grid sizes less than 1024 x and y access
 * 					The naieve call count is
 * 					read global memory calls: 		9
 * 					write global memory calls: 		2
 * 					read shared memory calls:		0
 * 					write shared memory calls:		0
 *
 * 					The optimized call count is
 * 					read global memory calls: 		3
 * 					write global memory calls: 		1
 * 					read shared memory calls:		12
 * 					write shared memory calls:		3
 *
 */
__global__ void gpuGameOfLifeOptimized(uint8_t* cellsGlobal, int64_t* dataX, int64_t* dataY, uint8_t *outputCellLocal)
{
	extern __shared__ uint8_t sData[];
	uint64_t tId = threadIdx.x + (blockIdx.x * blockDim.x);
	uint64_t x = *dataX, y = *dataY;
	uint64_t totalBlockCount = x*y;
	uint64_t yId = tId/y;
	uint64_t xId = tId - (yId*x);
	//Pass data to shared
	if(tId <= totalBlockCount)
	{
		uint8_t rowIndex = 0;
		uint64_t yVal = yCellMinus(1,yId,y);
		while (rowIndex < 3)
		{
			sData[(rowIndex * x) + xId] =cellsGlobal[((yVal*x)) + xId];
			yVal = yCellPlus(1,yVal, y);
			++rowIndex;
		}
	}
	syncthreads();
	if(tId <= totalBlockCount)
	{
		uint8_t localCellCount =  surrondingCellCountOptimized(sData, xId, x, y);
		uint64_t result = 0;
		switch(localCellCount)
		{
			case 2:
				if(sData[x + xId])
				{
					result = 1;
				}
				break;
			case 3:
				result = 1;
				break;
		}
		outputCellLocal[xId+(yId*x)] = result;
	}
}

__global__ void gpuGameOfLifeNaive(uint8_t* cellsLocal, int64_t* dataX, int64_t* dataY, uint8_t *outputCellLocal)
{

	uint64_t tId = threadIdx.x + (blockIdx.x * blockDim.x);
	uint64_t x = *dataX, y = *dataY;
	uint64_t totalBlockCount = x*y;
	if(tId <= totalBlockCount)
	{
		uint64_t yId = tId/y;
		uint64_t xId = tId - (yId*x);
		outputCellLocal[xId+(yId*x)] = 0;
		uint8_t localCellCount =  surrondingCellCount(cellsLocal, xId, yId, x, y);
		switch(localCellCount)
		{
			case 2:
				if(cellsLocal[xId+(yId*x)])
				{
					outputCellLocal[xId+(yId*x)] = 1;
				}
				break;
			case 3:
				outputCellLocal[xId+(yId*x)] = 1;
				break;
		}
	}
}

//Improved Naieve
//__global__ void gpuGameOfLifeNaive(uint8_t* cellsLocal, int64_t* dataX, int64_t* dataY, uint8_t *outputCellLocal)
//{
//	int64_t x = *dataX, y = *dataY;
//	int64_t xId = 0;
//	int64_t yIndex = threadIdx.x + (blockIdx.x * blockDim.x);
//	int64_t totalBlockCount = x*y;
//	if(yIndex <= totalBlockCount)
//	{
//		xId = 0;
//		while(xId < x)
//		{
//			outputCellLocal[xId+(yIndex*y)] = 0;
//			uint8_t localCellCount =  surrondingCellCount(cellsLocal, xId, yIndex, x, y);
//			switch(localCellCount)
//			{
//				case 2:
//					if(cellsLocal[xId+(yIndex*y)])
//					{
//						outputCellLocal[xId+(yIndex*y)] = 1;
//					}
//					break;
//				case 3:
//					outputCellLocal[xId+(yIndex*y)] = 1;
//					break;
//			}
//
//			++xId;
//		}
//	}
//}

//Origonal
//__global__ void gpuGameOfLifeNaive(uint8_t* cellsLocal, int64_t* dataX, int64_t* dataY, uint8_t *outputCellLocal)
//{
//	int64_t x = *dataX, y = *dataY;
//	int64_t yId =0, xId = 0;
//	int64_t ysId = threadIdx.x + (blockIdx.x * blockDim.x);
//	if(ysId == 1)
//	{
//	    while(yId < y)
//	    {
//	        xId = 0;
//	        while(xId < x)
//	        {
//	        	outputCellLocal[xId+(yId*y)] = 0;
//	            uint8_t localCellCount =  surrondingCellCount(cellsLocal, xId, yId, x, y);
//				switch(localCellCount)
//				{
//					case 2:
//						if(cellsLocal[xId+(yId*y)])
//						{
//							outputCellLocal[xId+(yId*y)] = 1;
//						}
//						break;
//					case 3:
//						outputCellLocal[xId+(yId*y)] = 1;
//						break;
//				}
//
//	            ++xId;
//	        }
//	        ++yId;
//	    }
//	}
//}
